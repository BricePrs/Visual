#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include "main.cuh"

#define CHECK_CUDA_ERROR() \
do { \
    hipError_t error = hipGetLastError(); \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s at %s:%d\n", hipGetErrorString(error), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

__global__
void saxpy(int n, float a, float *x, float *y)
{
    printf("lakz pakz ");
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) y[i] = a*x[i] + y[i];
}

int oui(void)
{
    int N = 1000;
    float *x, *y, *d_x, *d_y;
    x = (float *) malloc(N * sizeof(float));
    y = (float *) malloc(N * sizeof(float));

    hipMalloc(&d_x, N
                     *sizeof(float));
    hipMalloc(&d_y, N
                     *sizeof(float));

    for (
            int i = 0;
            i<N;
            i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N
                       *sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N
                       *sizeof(float), hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR();

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
    CHECK_CUDA_ERROR();

    hipMemcpy(y, d_y, N
                       *sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (
            int i = 0;
            i<N;
            i++)
        maxError = std::max(maxError, abs(y[i] - 4.0f));
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    return 0;
}

